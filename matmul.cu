#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <mma.h>

using namespace nvcuda;

#define M 512   // anything
#define N 513   // anything
#define K 514   // anything

// unset for no print final comparison
//#define DEBUG 1

// enable/disable CPU matmul
#define ENABLECPU 1

__global__ void matrixMultiplyV1(float* matrixA, float* matrixB,
                                    float* matrixC, float* matrixOut,
                                    float alpha,    float beta){
    
    int threadRowIndex      = blockIdx.x*blockDim.x + threadIdx.x;
    int threadColumnIndex   = blockIdx.y*blockDim.y + threadIdx.y;

    float tempSum = 0.0f;

    if (threadRowIndex < M){
        if (threadColumnIndex < N){
            // step through row[threadRowIndex] of matrixA
            // step through column[threadColumnIndex] of matrix B
            for (int step = 0; step < K; step++){
                tempSum += matrixA[threadRowIndex * K + step] * matrixB[step * N + threadColumnIndex];
            }

            tempSum *= alpha;

            // add matrixC[threadRowIndex][threadColumnIndex]
            tempSum += beta * matrixC[threadRowIndex * N + threadColumnIndex];

            // write to matrixOut
            matrixOut[threadRowIndex * N + threadColumnIndex] = tempSum;
        }
    }
}

__global__ void matrixMultiplyV2(float* matrixA, float* matrixB,
                                    float* matrixC, float* matrixOut,
                                    float alpha,    float beta){
    
    // swap these to coalesce global memory accesses
    int threadRowIndex      =   blockIdx.y*blockDim.y + threadIdx.y;
    int threadColumnIndex   =   blockIdx.x*blockDim.x + threadIdx.x;

    float tempSum = 0.0f;

    if (threadRowIndex < M){
        if (threadColumnIndex < N){
            // step through row[threadRowIndex] of matrixA
            // step through column[threadColumnIndex] of matrix B
            for (int step = 0; step < K; step++){
                tempSum += matrixA[threadRowIndex * K + step] * matrixB[step * N + threadColumnIndex];
            }

            tempSum *= alpha;

            // add matrixC[threadRowIndex][threadColumnIndex]
            tempSum += beta * matrixC[threadRowIndex * N + threadColumnIndex];

            // write to matrixOut
            matrixOut[threadRowIndex * N + threadColumnIndex] = tempSum;
        }
    }
}

__global__ void wmma_kernel(half *a, half *b, float *c) {
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
   wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

   // Initialize the output to zero
   wmma::fill_fragment(c_frag, 0.0f);

   // Load the inputs
   wmma::load_matrix_sync(a_frag, a, 16);
   wmma::load_matrix_sync(b_frag, b, 16);

   // Perform the matrix multiplication
   wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

   // Store the output
   wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}

__global__ void doNothingKernel(){
    return;
}

int main(){
    doNothingKernel<<<1,1>>>();
    printf("[M] = %d\n[N] = %d\n[K] = %d\n\n",M,N,K);

    double number_of_flops = (double)M*(double)N*(2.0*(double)K-1.0) + (double) M * (double) N;
    printf("Number of FLOPS: %lf\n", number_of_flops);

    float* matrixA;
    float* matrixB;
    float* matrixC;
    float* matrixOut;
    float alpha, beta;

    alpha = 3.0f;
    beta = 5.0f;

    matrixA     = (float*)malloc(sizeof(float) * M * K); printf("matrixA allocated with space for %d floats\n", M * K);
    matrixB     = (float*)malloc(sizeof(float) * K * N); printf("matrixB allocated with space for %d floats\n", K * N);
    matrixC     = (float*)malloc(sizeof(float) * M * N); printf("matrixC allocated with space for %d floats\n", M * N);
    matrixOut   = (float*)malloc(sizeof(float) * M * N); printf("matrixOut allocated with space for %d floats\n", M * N);

    srand(0);

    // intialise matrixA
    printf("Setting matrixA...\n");
    for (int i = 0; i < M; i++){
        for (int j = 0; j < K; j++){
            matrixA[i * K + j] = (float)rand()/(float)RAND_MAX;
            //printf("%f ", matrixA[i * K + j]);
        }
        //printf("\n");
    }

    // initialise matrixB
    printf("Setting matrixB...\n");
    for (int i = 0; i < K; i++){
        for (int j = 0; j < N; j++){
            matrixB[i * N + j] = (float)rand()/(float)RAND_MAX;
            //printf("%f ", matrixB[i * N + j]);
        }
        //printf("\n");
    }

    // initialise matrixC
    printf("Setting matrixC...\n");
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            matrixC[i * N + j] = (float)rand()/(float)RAND_MAX;
            //printf("%f ", matrixC[i * M + j]);
        }
        //printf("\n");
    }

    // set output to 0
    memset(matrixOut, 0, sizeof(float) * M * N);
    printf("matrixOut set to all zeroes\n");

    // do the matrix multiply

#ifdef ENABLECPU
    auto start_cpu_mmul = std::chrono::high_resolution_clock::now();

    // step through rows of the output
    for (int i = 0; i < M; i++){
        // step through columns of the output
        for (int j = 0; j < N; j++){
            float tempSum = 0.0f;
            for (int step = 0; step < K; step++){
                // step through row[i] of matrixA
                // step through column[j] of matrixB
                //printf("Accessing element %d of matrixA[%d] and element %d of matrixB[%d]\n", i*K + step, M * K, step * N + j, K * N);
                tempSum += matrixA[i*K + step] * matrixB[step * N + j] * alpha;
            }
            // add [i][j] of matrixC
            tempSum += beta * matrixC[i * N + j];
            matrixOut[i * N + j] = tempSum;
            //printf("Accessing element %d of matrixOut[%d]\n", i * N + j, M * N);
        }
    }

    auto stop_cpu_mmul = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_cpu_mmul = stop_cpu_mmul - start_cpu_mmul;
    printf("CPU MMUL took %f ms @ %lf GFLOPs/s\n", duration_cpu_mmul.count(), number_of_flops / ((double) duration_cpu_mmul.count()/1000) / 1024 / 1024 / 1024);
#endif

    float* GPU_matrixA;
    float* GPU_matrixB;
    float* GPU_matrixC;
    float* GPU_matrixOut;
    float* hostCheck_matrixOut;

    hipError_t errorCheckVariable;

    errorCheckVariable = hipMalloc((void**)&GPU_matrixA,   sizeof(float) * M * K);
    errorCheckVariable = hipMalloc((void**)&GPU_matrixB,   sizeof(float) * K * N);
    errorCheckVariable = hipMalloc((void**)&GPU_matrixC,   sizeof(float) * M * N);
    errorCheckVariable = hipMalloc((void**)&GPU_matrixOut, sizeof(float) * M * N);

    printf("\nSetting hostCheck_matrixOut to all zeroes\n");
    hostCheck_matrixOut = (float*)malloc(sizeof(float) * M * N);
    printf("\nSet hostCheck_matrixOut to all zeroes\n");

    errorCheckVariable = hipMemcpy(GPU_matrixA, matrixA,       sizeof(float) * M * K, hipMemcpyHostToDevice);
    errorCheckVariable = hipMemcpy(GPU_matrixB, matrixB,       sizeof(float) * K * N, hipMemcpyHostToDevice);
    errorCheckVariable = hipMemcpy(GPU_matrixC, matrixC,       sizeof(float) * M * N, hipMemcpyHostToDevice);
    errorCheckVariable = hipMemcpy(GPU_matrixOut, matrixOut,   sizeof(float) * M * N, hipMemcpyHostToDevice);

    int blockSizeX = 32;
    int blockSizeY = 32;
    dim3 blockSize =    dim3(blockSizeX,blockSizeY);

    int gridSizeX = ceil((double) N / (double) blockSizeX);
    int gridSizeY = ceil((double) M / (double) blockSizeY);

    dim3 gridSize =     dim3(gridSizeX,gridSizeY);

    printf("blockSize   = [%d,  %d]\n", blockSizeX, blockSizeY);
    printf("gridSize    = [%d,  %d]\n", gridSizeX, gridSizeY);

    hipEvent_t start_mmul, stop_mmul;
    float milliseconds_mmul, seconds_mmul;
    hipEventCreate(&start_mmul);
    hipEventCreate(&stop_mmul);

    double nRepeats = 10.0;

    // V1 Kernel
    hipEventRecord(start_mmul);
    for (double i = 0; i < nRepeats; i+=1.0) matrixMultiplyV1<<<gridSize,blockSize>>>(GPU_matrixA, GPU_matrixB, GPU_matrixC, GPU_matrixOut, alpha, beta);

    hipDeviceSynchronize();
    hipEventRecord(stop_mmul);
    hipEventSynchronize(stop_mmul);
    hipEventElapsedTime(&milliseconds_mmul, start_mmul, stop_mmul);
    seconds_mmul = milliseconds_mmul/ 1000;
    printf("GPU MMUL V1 took %f ms @ %lf GFLOPS/s\n", milliseconds_mmul, number_of_flops * nRepeats / (double) seconds_mmul / 1024 / 1024 / 1024 );

    
    // V2 Kernel
    hipEventRecord(start_mmul);

    for (double i = 0; i < nRepeats; i+=1.0) matrixMultiplyV2<<<gridSize,blockSize>>>(GPU_matrixA, GPU_matrixB, GPU_matrixC, GPU_matrixOut, alpha, beta);

    hipDeviceSynchronize();
    hipEventRecord(stop_mmul);
    hipEventSynchronize(stop_mmul);
    hipEventElapsedTime(&milliseconds_mmul, start_mmul, stop_mmul);
    seconds_mmul = milliseconds_mmul/ 1000;
    printf("GPU MMUL V2 took %f ms @ %lf GFLOPS/s\n", milliseconds_mmul, number_of_flops * nRepeats / (double) seconds_mmul / 1024 / 1024 / 1024 );

    // copy output back to host
    hipMemcpy(hostCheck_matrixOut, GPU_matrixOut, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    float maxDiscrepancy = 0.0f;
    float tempDiscrepancy = 0.0f;

    // print matrixOut
    printf("\nmatrixOut:\n");
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            tempDiscrepancy = matrixOut[i * N + j] - hostCheck_matrixOut[i * N + j];
#ifdef DEBUG
            printf("%f vs ", matrixOut[i * N + j]);
            printf("%f  ", hostCheck_matrixOut[i * N + j]);
#endif
            if (abs(tempDiscrepancy) > maxDiscrepancy){
                maxDiscrepancy = abs(tempDiscrepancy);
            }
        }
#ifdef DEBUG
        printf("\n");
#endif
    }

    printf("maxDiscrepancy = %f\n", maxDiscrepancy);
}